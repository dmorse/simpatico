#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_complex.h>

__global__ void kernel_calculate_sq_partial(
            int n_particles,
            hipComplex *fourier_mode_partial,
            float3 *pos,
            int n_wave,
            float3 *wave_vectors,
            float *d_modes, 
            int *d_type)
    {
    extern __shared__ hipComplex sdata[];

    unsigned int tidx = threadIdx.x;

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for (unsigned int i = 0; i < n_wave; i++) {
        float3 q = wave_vectors[i];

        hipComplex mySum = make_hipComplex(0.0,0.0);

        if (j < n_particles) {
            
            float3 p = pos[j];
            float dotproduct = q.x * p.x + q.y * p.y + q.z * p.z;
            int type = d_type[j];
            float mode = d_modes[type];
            hipComplex exponential = make_hipComplex(mode*cosf(dotproduct),
                                                   mode*sinf(dotproduct));
            mySum = hipCaddf(mySum,exponential);
        }
        sdata[tidx] = mySum;
    
       __syncthreads();

        // reduce in shared memory
        if (blockDim.x >= 512) {
           if (tidx < 256) {sdata[tidx] = mySum = hipCaddf(mySum,sdata[tidx+256]); }
            __syncthreads();
        }

        if (blockDim.x >= 256) {
           if (tidx < 128) {sdata[tidx] = mySum = hipCaddf(mySum, sdata[tidx + 128]); }
           __syncthreads(); 
        }

        if (blockDim.x >= 128) {
           if (tidx <  64) {sdata[tidx] = mySum = hipCaddf(mySum, sdata[tidx +  64]); }
           __syncthreads();
        }

        if (tidx < 32) {
            volatile hipComplex* smem = sdata;
            if (blockDim.x >= 64) {
                hipComplex rhs = hipCaddf(mySum, smem[tidx + 32]); 
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
            }
            if (blockDim.x >= 32) {
                hipComplex rhs = hipCaddf(mySum, smem[tidx + 16]); 
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
            }
            if (blockDim.x >= 16) {
                hipComplex rhs = hipCaddf(mySum, smem[tidx + 8]); 
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
            }
            if (blockDim.x >=  8) {
                hipComplex rhs = hipCaddf(mySum, smem[tidx + 4]); 
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
            }
            if (blockDim.x >=  4) {
                hipComplex rhs = hipCaddf(mySum, smem[tidx + 2]); 
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
            } 
            if (blockDim.x >=  2) {
                hipComplex rhs = hipCaddf(mySum, smem[tidx + 1]); 
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
            } 
        }

        // write result to global memeory
        if (tidx == 0)
           fourier_mode_partial[blockIdx.x + gridDim.x*i] = sdata[0];
    } // end loop over wave vectors
}

__global__ void kernel_calculate_norms(hipComplex* fourier_mode_partial,
                                       unsigned int nblocks, 
                                       float *sq_vec,
                                       int n_wave,
                                       float V)
    {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_wave)
        return;

    // do final reduction of fourier mode
    hipComplex fourier_mode = make_hipComplex(0.0,0.0);
    for (unsigned int j = 0; j < nblocks; j++)
       fourier_mode = hipCaddf(fourier_mode, fourier_mode_partial[j + i*nblocks]);
 
    float normsq = fourier_mode.x * fourier_mode.x + fourier_mode.y * fourier_mode.y;
    sq_vec[i] = normsq/V;
    }

int gpu_sample_structure_factor(int n_wave,
                                 float3 *h_wave_vectors,
                                 unsigned int n_particles,
                                 float3 *h_pos,
                                 int *h_types,
                                 int n_type,   
                                 int n_mode,
                                 float *h_modes,
                                 float *h_sq,
                                 float V
                                 ) 
    {
    float3* d_wave_vectors;
    float3* d_pos;
    int *d_type;
    float *d_modes;
    hipComplex *d_fourier_mode_partial;
    float *d_sq_vec;

    hipError_t cudaStatus;

    hipMalloc(&d_wave_vectors, sizeof(float3)*n_wave);
    hipMemcpy(d_wave_vectors, h_wave_vectors, sizeof(float3)*n_wave, hipMemcpyHostToDevice);

    hipMalloc(&d_pos, sizeof(float3)*n_particles);
    hipMemcpy(d_pos, h_pos, sizeof(float3)*n_particles, hipMemcpyHostToDevice);

    hipMalloc(&d_type, sizeof(int)*n_particles);
    hipMemcpy(d_type, h_types, sizeof(int)*n_particles, hipMemcpyHostToDevice);

    hipMalloc(&d_modes, sizeof(float)*n_type*n_mode);
    hipMemcpy(d_modes, h_modes, sizeof(float)*n_type*n_mode, hipMemcpyHostToDevice);

    const unsigned int block_size_x = 256;
    unsigned int n_blocks_x = n_particles/block_size_x + 1;

    hipMalloc(&d_fourier_mode_partial, sizeof(hipComplex)*n_wave*n_blocks_x);
    hipMalloc(&d_sq_vec, sizeof(float)*n_wave*n_mode);

    for (int i = 0; i < n_mode; i++)
        {
        unsigned int shared_size = block_size_x * sizeof(hipComplex);
        kernel_calculate_sq_partial<<<n_blocks_x, block_size_x, shared_size>>>(
               n_particles,
               d_fourier_mode_partial,
               d_pos,
               n_wave,
               d_wave_vectors,
               d_modes + i*n_type,
               d_type);
 
        if (cudaStatus = hipGetLastError()) {
               printf("CUDA ERROR (kernel_calculate_sq_partial): %s\n", hipGetErrorString(cudaStatus));
               return 1;
        }

        // calculate final S(q) values of this mode
        const unsigned int block_size = 512;
        kernel_calculate_norms<<<n_wave/block_size + 1, block_size>>>(d_fourier_mode_partial,
                                                                      n_blocks_x,
                                                                      d_sq_vec + i*n_wave,
                                                                      n_wave,
                                                                      V);

        if (cudaStatus = hipGetLastError())
            {
            printf("CUDA ERROR (kernel_calculate_norms): %s\n", hipGetErrorString(cudaStatus));
            return 1;
            }


        } // end loop over modes

    // copy back structure factors
    hipMemcpy(h_sq, d_sq_vec, n_wave*n_mode*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_wave_vectors);
    hipFree(d_pos);
    hipFree(d_type);
    hipFree(d_modes);
    hipFree(d_fourier_mode_partial);
    hipFree(d_sq_vec);

    return 0;
    }
